#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define BLOCK_NUM 8
#define BLOCK_SIZE 500

#define RANGE 19.87

/*** Declaration of the kernel function below this line ***/

__global__ void kernel(float *result);

/**** end of the kernel declaration ***/

int main(int argc, char *argv[])
{
  int i;                          // loop index
  float *result;                  // The arrays that will be processed in the host.
  float *resultd;                 // The arrays that will be processed in the device.
  struct timespec start, end;     // to meaure the time taken by a specific part of code
  int n = BLOCK_NUM * BLOCK_SIZE; // size of the array

  // Allocating the arrays in the host

  if (!(resultd = (float *)malloc(n * sizeof(float))))
  {
    printf("Error allocating array resultd\n");
    exit(1);
  }

  // Fill out the arrays with random numbers between 0 and RANGE;
  srand((unsigned int)time(NULL));
  for (i = 0; i < n; i++)
  {
    result[i] = 0.0;
  }

  /******************  The start GPU part: Do not modify anything in main() above this line  ************/
  // The GPU part

  // Allocate the arrays in the device
  hipMalloc((void **)&resultd, n * sizeof(float));

  // Copy the arrays from the host to the device
  hipMemcpy(resultd, result, n * sizeof(float), hipMemcpyHostToDevice);

  clock_gettime(CLOCK_REALTIME, &start);

  // Call the kernel function
  dim3 grid_size(BLOCK_NUM, 1, 1);
  dim3 block_size(BLOCK_SIZE, 1, 1);
  kernel<<<grid_size, block_size>>>(resultd);

  // Force host to wait on the completion of the kernel
  hipDeviceSynchronize();

  clock_gettime(CLOCK_REALTIME, &end);

  // Copy the result from the device to the host
  hipMemcpy(result, resultd, n * sizeof(float), hipMemcpyDeviceToHost);

  // Free the memory allocated in the device
  hipFree(resultd);

  printf("Total time taken by the GPU part = %lf\n", (double)(end.tv_sec - start.tv_sec) + (double)(end.tv_nsec - start.tv_nsec) / 1000000000);
  /******************  The end of the GPU part: Do not modify anything in main() below this line  ************/

  // Checking the correctness of the GPU part
  for (i = 0; i < n; i++)
    printf("Element %d: %f", i, result[i]);

  // Free the arrays in the host
  free(result);

  return 0;
}

/**** Write the kernel itself below this line *****/

__global__ void kernel(float *result)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int sharedMemeorySize = blockIdx.x % 32;
  __shared__ float sharedMemory[sharedMemeorySize];
  sharedMemory[threadIdx.x % sharedMemeorySize] += 1;

  __syncthreads();

  result[i] = sharedMemory[threadIdx.x % sharedMemeorySize];
}
